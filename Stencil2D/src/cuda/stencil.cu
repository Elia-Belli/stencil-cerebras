#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <chrono>
#include <fstream>
#include <sstream>
#include <vector>
#include <iomanip> // For setprecision
#include <hip/hip_runtime.h>

#include "utils.hpp"

using namespace std;

void getDeviceProperties(int device, hipDeviceProp_t* prop);
int getBlockSize(hipDeviceProp_t* prop, int threads, int sharedPerThread, int regsPerThread);

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}


__global__
void stencil(float* a, float* a_aux, int rows, int cols, float diff)
{

    int i, j;
    for(i = 1; i < rows-1; i++){
        for(j = 1; j < cols-1; j++){
            a_aux[i * cols + j] = a[(i-1) * cols + j]
                                + a[(i+1) * cols + j]
                                + a[i * cols + (j+1)]
                                + a[(i+1) * cols + (j-1)]
                                -diff * a[i * cols + j];
        }   
    }
}


int main(int argc, char* argv[])
{
    string input = "./inputs/10x10.csv";
    int rows = 10;
    int cols = 10;
    int iterations = 1;
    float diff = 4.0;
    
    int size = (rows+2) * (cols+2);
    float *a = (float*) calloc(size, sizeof(float));
    readCSV(input, a, rows, cols+2);
    printMatrix(a, rows+2, cols+2);

    // Device Info
    CHECK_CUDA_CALL( hipSetDevice(0) );
    CHECK_CUDA_CALL( hipDeviceSynchronize() );
    hipDeviceProp_t prop;
    getDeviceProperties(0, &prop);

    // Allocate GPU data structures
    float *d_a, *d_aux, *temp;
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_aux, size * sizeof(float));

    // Send data to GPU
    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    
    for(int i = 0; i < iterations; i++){

        stencil<<<1, 1, 1>>>(d_a, d_aux, rows+2, cols+2, diff);
        CHECK_CUDA_CALL(hipDeviceSynchronize());

        temp = d_a;
        d_a = d_aux;
        d_aux = temp;
    }

    // Copy result to CPU
    hipMemcpy(a, d_a, size * sizeof(float), hipMemcpyDeviceToHost);
    printMatrix(a, rows+2, cols+2);


    //Free GPU memory
    hipFree(d_a);
    hipFree(d_aux);

    // Free CPU memory
    free(a);

    return 0;
}


/*
    Gets properties of cuda device
    DEBUG MODE: prints properties
*/
void getDeviceProperties(int device, hipDeviceProp_t *prop)
{
    hipGetDeviceProperties(prop, device);

    #ifdef DEBUG
    printf("\nDevice %d Properties\n", device);
    printf("  Memory Clock Rate (MHz): %d\n", prop->memoryClockRate/1024);
    printf("  Memory Bus Width (bits): %d\n", prop->memoryBusWidth);

    printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
        2.0*prop->memoryClockRate*(prop->memoryBusWidth/8)/1.0e6);
    printf("  Total global memory (Gbytes) %.1f\n",(float)(prop->totalGlobalMem)/1024.0/1024.0/1024.0);
    printf("  Shared memory per block (Bytes) %.1f\n",(float)(prop->sharedMemPerBlock));
    printf("  Shared memory per SM (Bytes) %.1f\n",(float)(prop->sharedMemPerMultiprocessor));
    
    printf("  SM count : %d\n", prop->multiProcessorCount);
    printf("  Warp-size: %d\n", prop->warpSize);
    printf("  max-threads-per-block: %d\n", prop->maxThreadsPerBlock);
    printf("  max-threads-per-multiprocessor: %d\n", prop->maxThreadsPerMultiProcessor);
    printf("  register-per-block: %d\n", prop->regsPerBlock);
    #endif
    
}

/*
    Compute ideal blockSize for a kernel
*/
int getBlockSize(hipDeviceProp_t* prop, int threads, int sharedPerThread, int regsPerThread)
{

    int warpSize = prop->warpSize;
    int regsPerBlock = prop->regsPerBlock;
    int sharedMem = prop->sharedMemPerBlock;

    // For cc >= 3.0 we have at least 4 warpSchedulers per SM

    int blockSize = 4*warpSize;
    blockSize = min(blockSize, regsPerBlock/regsPerThread);
    blockSize = min(blockSize, sharedMem/sharedPerThread);
    blockSize = min(blockSize, prop->maxThreadsPerMultiProcessor);

    blockSize = warpSize * ceil(blockSize/warpSize);

    return blockSize;
}